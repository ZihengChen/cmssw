#include "hip/hip_runtime.h"
#include "RecoLocalCalo/HGCalRecProducers/interface/BinnerGPU.h"
#include "HeterogeneousCore/CUDAUtilities/interface/GPUVecArray.h"
#include <math.h>

#include "RecoLocalCalo/HGCalRecProducers/interface/GPUHist2D.h"


namespace BinnerGPU {


  __global__ void kernel_compute_histogram(RecHitGPU *dInputData, Histo2D *dOutputData, const size_t numRechits) {

    size_t rechitLocation = blockIdx.x * blockDim.x + threadIdx.x;

    if(rechitLocation >= numRechits)
        return;

    float x = dInputData[rechitLocation].x;
    float y = dInputData[rechitLocation].y;
   
    dOutputData->fillBinGPU(x, y, rechitLocation);

  }


  float minEta = 1.6;
  float maxEta = 3.0;
  float minPhi = -M_PI;
  float maxPhi = M_PI;

  float minX = -300.0, minY = -300.0;
  float maxX = 300.0, maxY = 300.0;

  Histo2D computeBins(std::vector<RecHitGPU> layerData) {
    Histo2D hOutputData(minX, maxX, minY, maxY);

    // Allocate memory and put data into device
    Histo2D *dOutputData;
    RecHitGPU* dInputData;
    hipMalloc(&dOutputData, sizeof(Histo2D));
    hipMalloc(&dInputData, sizeof(RecHitGPU)*layerData.size());
    hipMemcpy(dInputData, layerData.data(), sizeof(RecHitGPU)*layerData.size(), hipMemcpyHostToDevice);
    hipMemset(dOutputData, 0x00, sizeof(Histo2D));
    hipMemcpy(dOutputData, &hOutputData, sizeof(Histo2D), hipMemcpyHostToDevice);
    // Call the kernel
    const dim3 blockSize(1024,1,1);
    const dim3 gridSize(ceil(layerData.size()/1024.0),1,1);
    kernel_compute_histogram <<<gridSize,blockSize>>>(dInputData, dOutputData, layerData.size());

    // Copy result back!
    hipMemcpy(&hOutputData, dOutputData, sizeof(Histo2D), hipMemcpyDeviceToHost);
  
    // Free all the memory
    hipFree(dOutputData);
    hipFree(dInputData);

    
    return hOutputData;
  }

}
